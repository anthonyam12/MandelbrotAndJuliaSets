#include "hip/hip_runtime.h"
#include "global.h"

Julia_cu::Julia_cu()
{
	this->ComplexXMin = -2.0;
	this->ComplexXMax = 2.0;
	this->ComplexYMin = -2.0;
	this->ComplexYMax = 2.0;
}

__global__ void CalcPoint<<< nBlocks, nThreads >>>( float *d_x, float *d_y, float *d_scheme, 
									int nx, int ny, int maxIter, double seedx, double seedy )
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if ( index < nx )
	{
		int start = index*ny;
		int end = start + ny;
		for ( int i = start; i < end; i++ )
		{

			double zx = x[index];
			double zy = y[i];

			int count = 0;
			
			while( ( zx*zx + zy*zy <= 4.0 ) && ( count < maxIter ) )
			{
				// complex square
				double new_zx = zx*zx - zy*zy;
				double new_zy = 2 * zx * zy;
				zx = new_zx;
				zy = new_zy;

				// add z0
				zx += seedx;
				zy += seedy;

				// incr count
				count++;
			}
		
			// set color
			if( count >= maxIter )
			{
				scheme[i] = 0;
			}
			else if ( count > ( maxIter / 8) ) 
			{
				scheme[i] = 2;
			}
			else if ( count > ( maxIter / 10) ) 
			{
				scheme[i] = 3;
			}
			else if ( count > ( maxIter / 20) ) 
			{
				scheme[i] = 4;
			}
			else if ( count > ( maxIter / 40) ) 
			{
				scheme[i] = 5;
			}
			else if ( count > ( maxIter / 100) ) 
			{
				scheme[i] = 6;
			}
			else if ( count > (maxIter / 200) )
			{
				scheme[i] = 7;
			}
			else if ( count > (maxIter / 400) )
			{
				scheme[i] = 8;
			}
			else if ( count > (maxIter / 600) )
			{
				scheme[i] = 9;
			}
			else if ( count > (maxIter / 800) )
			{
				scheme[i] = 1;
			}
			else 
			{
				scheme[i] = 10;
			}
		}
	}
}


vector< ComplexPoint > Julia_cu::GetPoints( ComplexPoint seed, int nx, int ny,  int maxIters )
{
	vector< ComplexPoint > points;
	ComplexPoint zIncr, z;

	double seedX = seed.x;
	double seedY = seed.y;
	
	int size_nx = nx * sizeof( float );
	int size_nynx = ny * nx * sizeof( float );
	int size_sch = ny * nx * sizeof( int );

	float *x = ( float* )malloc(size_nx);
	float *y = ( float* )malloc(size_nynx);
	int *scheme = ( int* )malloc(size_sch);

	ComplexWidth = ComplexXMax - ComplexXMin;
	ComplexHeight = ComplexYMax - ComplexYMin;
	
	zIncr.x = ComplexWidth / float( nx );
	xIncr.y = ComplexHeight / float( ny );

	for( int i = 0; i < nx; i++ )
	{
		x[i] = ComplexXMin + ( xIncr.x * i );
		int multiplier = 0;
		for( int j = 0; j < nx; j++ )
		{
			y[j] = ComplexYMin + ( zIncr.y * multiplier );
			scheme[i] = 0;
			multiplier++;
		}
	}

	float *d_x, *d_y;
	int *d_scheme;

	hipMalloc( ( void** )&d_x, size_nx );
	hipMalloc( ( void** )&d_y, size_nynx );
	hipMalloc( ( void** )&d_scheme, size_sch );

	hipMemcpy( d_x, x, size_nx, hipMemcpyHostToDevice );
	hipMemcpy( d_y, y, size_nynx, hipMemcpyHostToDevice );
	hipMemcpy( d_scheme, scheme, size_nynx, hipMemcpyHostToDevice );

	int nThreads = 64;
	int nBlocks = ( nx + nThreds - 1 ) / nThreads;

	CalcPoint<<< nBlocks, nThreads >>>( d_x, d_y, d_scheme, nx, ny, maxIter, seedx, seedy );
	
	hipMemcpy( x, d_x, size_nx, hipMemcpyDeviceToHost );
	hipMemcpy( y, d_y, size_nynx, hipMemcpyDeviceToHost );
	hipMemcpy( scheme, d_scheme, size_nynx, hipMemcpyDeviceToHost );

	for( int i = 0; i < nx; i++ )
	{
		z.x = x[i];
		for ( int j = i*ny; j < (i+1)*ny; j++ )
		{
			z.y = y[j];
			z.schemeIndex = scheme[j]l
			points.push_back( z );
		}
	}

	free(x); free(y); free(scheme);
	hipFree(d_x); hipFree(d_y); hipFree(d_scheme);
	return points;
}

double Julia_cu::GetComplexXMin()
{
	return ComplexXMin;
}

double Julia_cu::GetComplexXMax() 
{
	return ComplexXMax;
}

double Julia_cu::GetComplexYMin()
{
	return ComplexYMin;
}

double Julia_cu::GetComplexYMax() 
{
	return ComplexYMax;
}

void Julia_cu::SetComplexXMin( double xmin )
{
	this->ComplexXMin = xmin;
}


void Julia_cu::SetComplexXMax( double xmax )
{
	this->ComplexXMax = xmax;
}

void Julia_cu::SetComplexYMin( double ymin )
{
	this->ComplexYMin = ymin;
}

void Julia_cu::SetComplexYMax( double ymax )
{
	this->ComplexYMax = ymax;
}
