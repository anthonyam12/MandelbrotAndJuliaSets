#include "hip/hip_runtime.h"
#include "global.h"

Julia_cu::Julia_cu()
{
	// sets the default xmin, xmax, ymin, and ymax sets the julia set 
	// points are in this range
	this->ComplexXMin = -2.0;
	this->ComplexXMax = 2.0;
	this->ComplexYMin = -2.0;
	this->ComplexYMax = 2.0;
}

// Ran on the GPU, sets the scheme[] value for each points. This value is used
// as an index into the ColorScheme class to determine the color of a point
__global__ void CalcPoint( float *x, float *y, int *scheme, int nx, 
						   int ny, int maxIter, double seedx, double seedy )
{
	// get the index
	int index = threadIdx.x + blockIdx.x * blockDim.x;

	// determine if we need to run this index (in our range)
	if ( index < nx )
	{
		// start and end position in the y array
		int start = index*ny;
		int end = start + ny;

		// iterate the columns for this row
		for ( int i = start; i < end; i++ )
		{
			// get this x + yi point
			double zx = x[index];
			double zy = y[i];

			int count = 0;
			
			// determine convergence/divergence
			while( ( zx*zx + zy*zy <= 4.0 ) && ( count < maxIter ) )
			{
				// complex square
				double new_zx = zx*zx - zy*zy;
				double new_zy = 2 * zx * zy;
				zx = new_zx;
				zy = new_zy;

				// add z0
				zx += seedx;
				zy += seedy;

				// incr count
				count++;
			}
		
			// set color
			if( count >= maxIter )
			{
				scheme[i] = 0;
			}
			else if ( count > ( maxIter / 8) ) 
			{
				scheme[i] = 2;
			}
			else if ( count > ( maxIter / 10) ) 
			{
				scheme[i] = 3;
			}
			else if ( count > ( maxIter / 20) ) 
			{
				scheme[i] = 4;
			}
			else if ( count > ( maxIter / 40) ) 
			{
				scheme[i] = 5;
			}
			else if ( count > ( maxIter / 100) ) 
			{
				scheme[i] = 6;
			}
			else if ( count > (maxIter / 200) )
			{
				scheme[i] = 7;
			}
			else if ( count > (maxIter / 400) )
			{
				scheme[i] = 8;
			}
			else if ( count > (maxIter / 600) )
			{
				scheme[i] = 9;
			}
			else if ( count > (maxIter / 800) )
			{
				scheme[i] = 1;
			}
			else 
			{
				scheme[i] = 10;
			}
		}
	}
}


vector< ComplexPoint > Julia_cu::GetPoints( ComplexPoint seed, int nx, int ny,  int maxIter )
{
	vector< ComplexPoint > points;
	ComplexPoint zIncr, z;
	
	// get size of the arrays
	int size_nx = nx * sizeof( float );
	int size_nynx = ny * nx * sizeof( float );
	int size_sch = ny * nx * sizeof( int );

	// create x,y point arrays and a scheme array to store indexes into color scheme class
	float *x = ( float* )malloc(size_nx);
	float *y = ( float* )malloc(size_nynx);
	int *scheme = ( int* )malloc(size_sch);

	// determine the width and height of the screen
	ComplexWidth = ComplexXMax - ComplexXMin;
	ComplexHeight = ComplexYMax - ComplexYMin;
	
	// determine the increments in x and y
	zIncr.x = ComplexWidth / float( nx );
	zIncr.y = ComplexHeight / float( ny );

	// fill the x and y arrays with each x and y point we are working with
	for( int i = 0; i < nx; i++ )
	{
		x[i] = ComplexXMin + ( zIncr.x * i );
		int multiplier = 0;
		for( int j = i*ny; j < (i+1)*ny; j++ )
		{
			y[j] = ComplexYMin + ( zIncr.y * multiplier );
			scheme[i] = 0;
			multiplier++;
		}
	}

	// create the device x,y, and scheme arrays
	float *d_x, *d_y;
	int *d_scheme;

	// allocate the device memory
	hipMalloc( ( void** )&d_x, size_nx );
	hipMalloc( ( void** )&d_y, size_nynx );
	hipMalloc( ( void** )&d_scheme, size_sch );

	// copy the arrays to the device
	hipMemcpy( d_x, x, size_nx, hipMemcpyHostToDevice );
	hipMemcpy( d_y, y, size_nynx, hipMemcpyHostToDevice );
	hipMemcpy( d_scheme, scheme, size_nynx, hipMemcpyHostToDevice );

	// set number of blocks and threads on GPU
	int nThreads = 64;
	int nBlocks = ( nx + nThreads - 1 ) / nThreads;

	// calculate points on the GPU
	CalcPoint<<< nBlocks, nThreads >>>( d_x, d_y, d_scheme, nx, ny, maxIter, seed.x, seed.y );
	
	// copy memory back to host
	hipMemcpy( x, d_x, size_nx, hipMemcpyDeviceToHost );
	hipMemcpy( y, d_y, size_nynx, hipMemcpyDeviceToHost );
	hipMemcpy( scheme, d_scheme, size_nynx, hipMemcpyDeviceToHost );

	// fill the point vector with x,y points and the scheme indexes
	for( int i = 0; i < nx; i++ )
	{
		z.x = x[i];
		for ( int j = i*ny; j < (i+1)*ny; j++ )
		{
			z.y = y[j];
			z.schemeIndex = scheme[j];
			points.push_back( z );
		}
	}

	// free the memory
	free(x); free(y); free(scheme);
	hipFree(d_x); hipFree(d_y); hipFree(d_scheme);
	return points;
}

// getters and setters for the xmin, xmax, ymin, and ymax
double Julia_cu::GetComplexXMin()
{
	return ComplexXMin;
}

double Julia_cu::GetComplexXMax() 
{
	return ComplexXMax;
}

double Julia_cu::GetComplexYMin()
{
	return ComplexYMin;
}

double Julia_cu::GetComplexYMax() 
{
	return ComplexYMax;
}

void Julia_cu::SetComplexXMin( double xmin )
{
	this->ComplexXMin = xmin;
}

void Julia_cu::SetComplexXMax( double xmax )
{
	this->ComplexXMax = xmax;
}

void Julia_cu::SetComplexYMin( double ymin )
{
	this->ComplexYMin = ymin;
}

void Julia_cu::SetComplexYMax( double ymax )
{
	this->ComplexYMax = ymax;
}
