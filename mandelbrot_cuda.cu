#include "hip/hip_runtime.h"
#include "global.h"

Mandelbrot_cu::Mandelbrot_cu()
{
		this->ComplexXMin = -2.00;
		this->ComplexXMax = .5;
		this->ComplexYMin = -1.25;
		this->ComplexYMax = 1.25;

		this->ComplexWidth = ComplexXMax - ComplexXMin;
		this->ComplexHeight = ComplexYMax - ComplexYMin;
}

__global__ void CalcPoint( float *x, float *y, float *r, float *g, float *b, int w, int h, int n ) 
{
	for ( int index = blockIdx.x; index < n; index += gridDim.x )
	{
		
	}
}

vector< ComplexPoint > Mandelbrot_cu::GetPoints( int nx, int ny, int maxIter, Color colorScheme[7] ) 
{
	//TODO: Get the colors for the points from colorScheme[i] rather than setting to the book color
	//TODO: for animation we will go through our stored colored schemes and set the color in the 
	//TODO: MandelbrotPoints vector rather than calling this metod. As a matter of fact, it might
	//TODO: be best change the Color in the point struct to an int colorIndex; which would be an 
	//TODO: index to the color scheme we are currently using (values 0-6). Then we would just reset
	//TODO: a global colorScheme[7] array's values to the next color scheme and redraw the points.
	//TODO: We wouldn't need to change the points' colors at all since it is just an index to our 
	//TODO: current color scheme.
	
	vector< ComplexPoint > points;
	/*
	Color ptColor;

	ComplexPoint z, zIncr;
	int count = 0;

	zIncr.x = ComplexWidth / float( nx );
	zIncr.y = ComplexHeight / float( ny );

	for( z.x = ComplexXMin; z.x < ComplexXMax; z.x += zIncr.x )
	{
		for( z.y = ComplexYMin; z.y < ComplexYMax; z.y += zIncr.y ) 
		{
			//TODO: should we use more subdivisions for more definition?
			count = MandelbrotSqTransf( z, maxIter );
			if( count >= maxIter )
			{
				ptColor.r = ptColor.g = ptColor.b = 0.0;
			}
			else if ( count > ( maxIter / 8) ) 
			{
				ptColor.r = 1.0;
				ptColor.g = 0.5;
				ptColor.b = 0.0;
			}
			else if ( count > ( maxIter / 10) ) 
			{
				ptColor.r = 1.0;
				ptColor.g = 0.0;
				ptColor.b = 0.0;
			}
			else if ( count > ( maxIter / 20) ) 
			{
				ptColor.r = 0.0;
				ptColor.g = 0.0;
				ptColor.b = 0.5;
			}
			else if ( count > ( maxIter / 40) ) 
			{
				ptColor.r = 1.0;
				ptColor.g = 1.0;
				ptColor.b = 0.0;
			}
			else if ( count > ( maxIter / 100) ) 
			{
				ptColor.r = 0.0;
				ptColor.g = 0.3;
				ptColor.b = 0.0;
			}
			else 
			{
				ptColor.r = 0.0;
				ptColor.g = ptColor.b = 1.0;
			}
			z.color = ptColor;
			points.push_back(z);
		}
	}

	return points;
	*/
	long size = nx * ny;
	float *x, *y, *r, *g, *b;

	hipMalloc( ( void ** )&x, size );
	hipMalloc( ( void ** )&y, size );
	hipMalloc( ( void ** )&r, size );
	hipMalloc( ( void ** )&g, size );
	hipMalloc( ( void ** )&b, size );
	
	int nThreads = 64;
	int nBlocks = ( size + nThreads - 1 ) / nThreads;
	
	CalcPoint<<< nBlocks, nThreads >>>( x, y, r, g, b, ComplexWidth, ComplexHeight, size );

	return points;
}

ComplexPoint Mandelbrot_cu::ComplexSquare( ComplexPoint z )
{
	ComplexPoint square;

	square.x = z.x * z.x - z.y * z.y;
	square.y = 2 * z.x * z.y;

	return square;
}

int Mandelbrot_cu::MandelbrotSqTransf( ComplexPoint z0, int maxIter ) 
{
	ComplexPoint z = z0;
	int count = 0;

	while ( ( z.x * z.x + z.y * z.y <= 4.0 ) && ( count < maxIter ) )
	{
		z = ComplexSquare( z );
		z.x += z0.x;
		z.y += z0.y;
		count++;
	}
	return count;
}


float Mandelbrot_cu::GetComplexXMin()
{
	return this->ComplexXMin;
}

float Mandelbrot_cu::GetComplexXMax()
{
	return this->ComplexXMax;
}

float Mandelbrot_cu::GetComplexYMin()
{
	return this->ComplexYMin;
}

float Mandelbrot_cu::GetComplexYMax()
{
	return this->ComplexYMax;
}


void Mandelbrot_cu::SetComplexXMin( float xmin )
{
	this->ComplexXMin = xmin;
}

void Mandelbrot_cu::SetComplexXMax( float xmax )
{
	this->ComplexXMax = xmax;
}

void Mandelbrot_cu::SetComplexYMin( float ymin )
{
	this->ComplexYMin = ymin;
}

void Mandelbrot_cu::SetComplexYMax( float ymax )
{
	this->ComplexYMax = ymax;
}
