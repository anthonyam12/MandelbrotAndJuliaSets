#include "hip/hip_runtime.h"
#include "global.h"

Mandelbrot_cu::Mandelbrot_cu()
{
		this->ComplexXMin = -2.00;
		this->ComplexXMax = 1;
		this->ComplexYMin = -1.5;
		this->ComplexYMax = 1.5;
}

__global__ void CalcPoint( float *x, float *y, int *scheme, int nx, int ny, int maxIter ) 
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if ( index < nx ) 
	{
		for( int i = 0; i < ny; i++ )
		{
			float zx0 = x[index];
			float zy0 = y[index + i];

			float zx = x[index];
			float zy = y[index + i];

			int count = 0;
			while( ( zx*zx + zy*zy <= 4.0 ) && ( count < maxIter ) )
			{
				// complex square
				float new_zx = zx*zx - zy*zy;
				float new_zy = 2 * zx * zy;
				zx = new_zx;
				zy = new_zy;

				// add z0
				zx += zx0;
				zy += zy0;

				// incr count
				count++;
			}

			// set color
			if( count >= maxIter )
			{
				scheme[index + i] = 0;
			}
			else if ( count > ( maxIter / 8) ) 
			{
				scheme[index + i] = 2;
			}
			else if ( count > ( maxIter / 10) ) 
			{
				scheme[index + i] = 3;
			}
			else if ( count > ( maxIter / 20) ) 
			{
				scheme[index + i] = 4;
			}
			else if ( count > ( maxIter / 40) ) 
			{
				scheme[index + i] = 5;
			}
			else if ( count > ( maxIter / 100) ) 
			{
				scheme[index + i] = 6;
			}
			else if ( count > (maxIter / 200) )
			{
				scheme[index + i] = 7;
			}
			else if ( count > (maxIter / 400) )
			{
				scheme[index + i] = 8;
			}
			else if ( count > (maxIter / 600) )
			{
				scheme[index + i] = 9;
			}
			else if ( count > (maxIter / 800) )
			{
				scheme[index + i] = 1;
			}
			else 
			{
				scheme[index + i] = 10;
			}
		}
	}
}

vector< ComplexPoint > Mandelbrot_cu::GetPoints( int nx, int ny, int maxIter ) 
{
	vector< ComplexPoint > points;

	int size_nx = nx * sizeof( float );
	int size_nynx = ( nx*ny ) * sizeof( float );
	int size_sch = ( nx*ny ) * sizeof( int );	

	float *x = ( float * )malloc(size_nx);
	float *y = ( float * )malloc(size_nynx);
	int *scheme = ( int *)malloc(size_sch);
	
	// fill arrays with points before passing
	ComplexPoint z, zIncr;
	
	ComplexWidth = ComplexXMax - ComplexXMin;
	ComplexHeight = ComplexYMax - ComplexYMin;

	zIncr.x = ComplexWidth / double( nx );
	zIncr.y = ComplexHeight / double( ny );

	for( int i = 0; i < nx; i++ )
	{
		// get and set complex x value
		x[i] = ComplexXMin + ( zIncr.x * i );
		for( int j = 0; j < ny; j++ )
		{
			// get and set complex y value (and default scheme)
			y[j+i] = ComplexYMin + ( zIncr.y * j);
			scheme[j] = 0;
		}
	}

	// Do host side CUDA prep and run kernel on CUDA device
	float *d_x, *d_y;
	int *d_scheme;
	hipMalloc( ( void** )&d_x, size_nx );
	hipMalloc( ( void** )&d_y, size_nynx );
	hipMalloc( ( void** )&d_scheme, size_sch );

	hipMemcpy( d_x, x, size_nx, hipMemcpyHostToDevice );
	hipMemcpy( d_y, y, size_nynx, hipMemcpyHostToDevice );
	hipMemcpy( d_scheme, scheme, size_sch,  hipMemcpyHostToDevice );

	int nThreads = 64;
	int nBlocks = ( nx + nThreads - 1 ) / nThreads;

	CalcPoint<<< nBlocks, nThreads >>>( d_x, d_y, d_scheme, nx, ny, maxIter );

	hipMemcpy( x, d_x, size_nx, hipMemcpyDeviceToHost );
	hipMemcpy( y, d_y, size_nynx, hipMemcpyDeviceToHost );
	hipMemcpy( scheme, d_scheme, size_sch, hipMemcpyDeviceToHost );


	for( int i = 0; i < nx; i++ )
	{
		z.x = x[i];
		for( int j = 0; j < ny; j++ )
		{
			z.y = y[i + j];
			z.schemeIndex = scheme[i + j];
			points.push_back(z);
		}
	}
	
	free(x); free(y); free(scheme);
	hipFree(d_x); hipFree(d_y); hipFree(d_scheme);
	return points;
}

double Mandelbrot_cu::GetComplexXMin()
{
	return this->ComplexXMin;
}

double Mandelbrot_cu::GetComplexXMax()
{
	return this->ComplexXMax;
}

double Mandelbrot_cu::GetComplexYMin()
{
	return this->ComplexYMin;
}

double Mandelbrot_cu::GetComplexYMax()
{
	return this->ComplexYMax;
}


void Mandelbrot_cu::SetComplexXMin( double xmin )
{
	this->ComplexXMin = xmin;
}

void Mandelbrot_cu::SetComplexXMax( double xmax )
{
	this->ComplexXMax = xmax;
}

void Mandelbrot_cu::SetComplexYMin( double ymin )
{
	this->ComplexYMin = ymin;
}

void Mandelbrot_cu::SetComplexYMax( double ymax )
{
	this->ComplexYMax = ymax;
}
